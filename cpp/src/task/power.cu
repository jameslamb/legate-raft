#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <math.h>

#include "legate_library.h"
#include "legate_raft_cffi.h"

#include <common/gpu_task_context.hpp>

#include "legate/utilities/dispatch.h"

namespace legate_raft {

namespace {

struct power_fn_cpu {
  template <legate::Type::Code CODE, int32_t DIM>
  void operator()(legate::PhysicalStore output, legate::PhysicalStore input, legate::Scalar value)
  {
    using VAL = legate::type_of_t<CODE>;

    auto shape = input.shape<DIM>();

    if (shape.empty()) return;

    auto input_acc  = input.read_accessor<VAL, DIM>();
    auto output_acc = output.write_accessor<VAL, DIM>();

    for (legate::PointInRectIterator<DIM> it(shape, false /*fortran order*/); it.valid(); ++it) {
      auto p        = *it;
      output_acc[p] = pow(input_acc[p], value.value<VAL>());
    }
  }
};

template <legate::Type::Code CODE>
constexpr bool is_supported_gpu =
  (CODE == legate::Type::Code::FLOAT32 || CODE == legate::Type::Code::FLOAT64);

template <typename value_t>
__global__ void power_kernel(value_t* out, const value_t* in, value_t value, size_t volume)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= volume) return;
  out[idx] = pow(in[idx], value);
}

template __global__ void power_kernel(float*, const float*, float, size_t);
template __global__ void power_kernel(double*, const double*, double, size_t);

struct power_fn_gpu {
  template <legate::Type::Code CODE,
            int32_t DIM,
            std::enable_if_t<is_supported_gpu<CODE>>* = nullptr>
  void operator()(legate::TaskContext& ctx,
                  legate::PhysicalStore output,
                  legate::PhysicalStore input,
                  legate::Scalar value)
  {
    using VAL = legate::type_of_t<CODE>;

    legate_raft::GPUTaskContext gpu_task_context{ctx};
    auto handle = gpu_task_context.handle();
    auto stream = handle.get_stream();

    auto shape = input.shape<DIM>();

    if (shape.empty()) return;

    auto volume = shape.volume();

    int block_size = 256;  // TODO: tune
    int num_blocks = (volume + block_size - 1) / block_size;

    auto input_acc  = input.read_accessor<VAL, DIM>();
    auto output_acc = output.write_accessor<VAL, DIM>();

    power_kernel<<<num_blocks, block_size, 0, stream>>>(
      output_acc.ptr(shape), input_acc.ptr(shape), value.value<VAL>(), volume);
    handle.sync_stream();
  }

  template <legate::Type::Code CODE,
            int32_t DIM,
            std::enable_if_t<!is_supported_gpu<CODE>>* = nullptr>
  void operator()(legate::TaskContext& ctx,
                  legate::PhysicalStore output,
                  legate::PhysicalStore input,
                  legate::Scalar value)
  {
    LEGATE_ABORT("Code type not supported");
  }
};

}  // namespace

class PowerTask : public Task<PowerTask, POWER> {
 public:
  static void cpu_variant(legate::TaskContext context)
  {
    auto input  = context.input(0);
    auto value  = context.scalar(0);
    auto output = context.output(0);

    legate::double_dispatch(
      input.data().dim(), input.data().code(), power_fn_cpu{}, output, input, value);
  }

  static void gpu_variant(legate::TaskContext context)
  {
    auto input  = context.input(0);
    auto value  = context.scalar(0);
    auto output = context.output(0);

    legate::double_dispatch(
      input.data().dim(), input.data().code(), power_fn_gpu{}, context, output, input, value);
  }
};

}  // namespace legate_raft

namespace {

static void __attribute__((constructor)) register_tasks()
{
  legate_raft::PowerTask::register_variants();
}

}  // namespace
