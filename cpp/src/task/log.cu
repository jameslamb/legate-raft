#include "hip/hip_runtime.h"
/* Copyright 2023 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <cmath>

#include "legate_library.h"
#include "legate_raft_cffi.h"
#include "pitches.h"

#include <common/gpu_task_context.hpp>

#include "legate/utilities/dispatch.h"

#include <raft/core/device_mdarray.hpp>
#include <raft/core/handle.hpp>
#include <raft/core/math.hpp>
#include <raft/linalg/map.cuh>

namespace legate_raft {

namespace {

struct log_fn_cpu {
  template <legate::Type::Code CODE, int32_t DIM>
  void operator()(legate::PhysicalStore output, legate::PhysicalStore input)
  {
    using VAL = legate::type_of_t<CODE>;

    auto shape = input.shape<DIM>();

    if (shape.empty()) return;

    auto input_acc  = input.read_accessor<VAL, DIM>();
    auto output_acc = output.write_accessor<VAL, DIM>();

    for (legate::PointInRectIterator<DIM> it(shape, false /*fortran order*/); it.valid(); ++it) {
      auto p        = *it;
      output_acc[p] = log(input_acc[p]);
    }
  }
};

template <legate::Type::Code CODE>
constexpr bool is_supported_gpu =
  (CODE == legate::Type::Code::FLOAT32 || CODE == legate::Type::Code::FLOAT64);

struct log_op {
  template <typename Type, typename... UnusedArgs>
  RAFT_INLINE_FUNCTION auto operator()(const Type& in, UnusedArgs...) const
  {
    return raft::log(in);
  }
};

template <typename wo_t, typename ro_t, typename shape_t, typename pitches_t>
__global__ void log_generic_kernel(wo_t out, ro_t in, shape_t shape, pitches_t pitches)
{
  int idx     = blockIdx.x * blockDim.x + threadIdx.x;
  auto volume = shape.volume();
  if (idx >= volume) return;
  auto point = pitches.unflatten(idx, shape.lo);
  out[point] = logf(in[point]);
}

struct log_fn_gpu {
  template <legate::Type::Code CODE,
            int32_t DIM,
            std::enable_if_t<is_supported_gpu<CODE>>* = nullptr>
  void operator()(legate::TaskContext& ctx,
                  legate::PhysicalStore output,
                  legate::PhysicalStore input)
  {
    using VAL = legate::type_of_t<CODE>;

    legate_raft::GPUTaskContext gpu_task_context{ctx};
    auto handle = gpu_task_context.handle();
    auto stream = handle.get_stream();

    auto shape = input.shape<DIM>();

    if (shape.empty()) return;

    auto output_acc = output.write_accessor<VAL, DIM>();
    auto input_acc  = input.read_accessor<VAL, DIM>();

    if (output_acc.accessor.is_dense_row_major(shape) and
        input_acc.accessor.is_dense_row_major(shape)) {
      auto output_view = raft::make_device_vector_view(output_acc.ptr(shape), shape.volume());
      auto input_view  = raft::make_device_vector_view(input_acc.ptr(shape), shape.volume());
      raft::linalg::map(handle, output_view, log_op{}, input_view);
    } else {
      Pitches<DIM - 1> pitches;
      auto volume = pitches.flatten(shape);

      int block_size = 256;  // TODO: tune
      int num_blocks = (volume + block_size - 1) / block_size;
      log_generic_kernel<<<num_blocks, block_size, 0, stream>>>(
        output_acc, input_acc, shape, pitches);
    }

    handle.sync_stream();
  }

  template <legate::Type::Code CODE,
            int32_t DIM,
            std::enable_if_t<!is_supported_gpu<CODE>>* = nullptr>
  void operator()(legate::TaskContext& ctx,
                  legate::PhysicalStore output,
                  legate::PhysicalStore input)
  {
    LEGATE_ABORT("Code type not supported");
  }
};

}  // namespace

class LogTask : public Task<LogTask, LOG> {
 public:
  static void cpu_variant(legate::TaskContext context)
  {
    auto input  = context.input(0);
    auto output = context.output(0);

    legate::double_dispatch(input.data().dim(), input.data().code(), log_fn_cpu{}, output, input);
  }

  static void gpu_variant(legate::TaskContext context)
  {
    auto input  = context.input(0);
    auto output = context.output(0);

    legate::double_dispatch(
      input.data().dim(), input.data().code(), log_fn_gpu{}, context, output, input);
  }
};

}  // namespace legate_raft

namespace {

static void __attribute__((constructor)) register_tasks()
{
  legate_raft::LogTask::register_variants();
}

}  // namespace
